#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cv.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace cv;

__global__ unsigned char ajustar(int valor)
{
	if(valor < 0)
	{
		valor = 0;
	}
	else if(valor > 255)
	{
		valor = 255;
	}
    return (unsigned char)valor;
}

__global__ void filtro_sobel(unsigned char *origen, unsigned char *destino, char *kernel, int alto, int ancho)
{
	int i,j,x,y;
	int valor = 0;
	unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int fil = blockIdx.y*blockDim.y+threadIdx.y;
	
	for(i = 0; i <= 2; i++)
	{
		for(j = 0; j <= 2; j++)
		{
			x=col + i - 1;
			y=fil + j - 1;
			if(x >= 0 && y >= 0 && x < ancho && y < alto)
			{
				valor += kernel[j*3+i]*origen[y*ancho+x];
			}
		}
	}
	destino[fil*ancho+col] = ajustar(valor);
}

int main(int argc, char **argv)
{
	int alto, ancho, tamanno;
	char cpu_kernel[] = {-1,0,1,-2,0,2,-1,0,1}
	char *gpu_kernel;
	char *nombre_imagen = argv[1];
	char *nombre_resultado;
	nombre_resultado = (char*)malloc(sizeof(char)*255);
	unsigned char *cpu_origen, *cpu_destino;
	unsigned char *gpu_origen, *gpu_destino;
	struct timeval inicio, fin; 
	double tiempo;

	Mat imagen;
	imagen = imread(nombre_imagen, CV_LOAD_IMAGE_GRAYSCALE);
	Size t_imagen = imagen.size();
	alto = t_imagen.height;
	ancho = t_imagen.width;
	tamanno = sizeof(unsigned char)*alto*ancho;

	cpu_origen = (unsigned char*)malloc(tamanno);
	cpu_destino = (unsigned char*)malloc(tamanno);
	hipMalloc((void**)&gpu_kernel,sizeof(char)*9);
	hipMalloc((void**)&gpu_origen,tamanno);
	hipMalloc((void**)&gpu_destino,tamanno);
	cpu_origen = imagen.data;

	gettimeofday(&inicio, NULL);
	
	hipMemcpy(gpu_origen,cpu_origen,tamanno, hipMemcpyHostToDevice);
	hipMemcpy(gpu_kernel,cpu_kernel,sizeof(char)*9, hipMemcpyHostToDevice);

	int t_bloque = 32;
	dim3 dim_bloque(t_bloque,t_bloque,1);
	dim3 dim_rejilla(ceil(ancho/float(t_bloque)),ceil(alto/float(t_bloque)),1);
	filtro_sobel<<<dim_rejilla,dim_bloque>>>(gpu_origen, gpu_destino, gpu_kernel, alto, ancho);
	hipDeviceSynchronize();

	hipMemcpy(cpu_destino,gpu_destino,tamanno, hipMemcpyDeviceToHost);

	gettimeofday(&fin, NULL);
	tiempo = ((fin.tv_sec  - inicio.tv_sec) * 1000000u + fin.tv_usec - inicio.tv_usec) / 1.e6;
	printf("%f\n",tiempo);		

	Mat resultado;
	resultado.create(alto,ancho,CV_8UC1);
	resultado.data = cpu_destino;
	nombre_resultado = strcat(nombre_imagen,".sobel_global.jpg");
	imwrite("./sobel_global.jpg",resultado);
	
	free(cpu_origen);
	free(cpu_destino);
	hipFree(gpu_kernel);
	hipFree(gpu_origen);
	hipFree(gpu_destino);
	
	return 0;
}