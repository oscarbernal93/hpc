#include "hip/hip_runtime.h"
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <cv.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace cv;

__constant__ char gpu_kernel[9];


__device__ unsigned char ajustar(int valor)
{
	if(valor < 0)
	{
		valor = 0;
	}
	else if(valor > 255)
	{
		valor = 255;
	}
    return (unsigned char)valor;
}

__global__ void filtro_sobel(unsigned char *origen, unsigned char *destino, int alto, int ancho)
{
	int i,j,x,y;
	int valor = 0;
	unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int fil = blockIdx.y*blockDim.y+threadIdx.y;
	
	for(i = 0; i <= 2; i++)
	{
		for(j = 0; j <= 2; j++)
		{
			x=col + i - 1;
			y=fil + j - 1;
			if(x >= 0 && y >= 0 && x < ancho && y < alto)
			{
				valor += (gpu_kernel[j*3+i])*(origen[y*ancho+x]);
			}
		}
	}
	destino[fil*ancho+col] = ajustar(valor);
}

int main(int argc, char **argv)
{
	hipError_t mi_error = hipSuccess;
	int alto, ancho, tamanno;
	char cpu_kernel[] = {-1,0,1,-2,0,2,-1,0,1};
	char *nombre_imagen = argv[1];
	char *nombre_resultado;
	nombre_resultado = (char*)malloc(sizeof(char)*255);
	unsigned char *cpu_origen, *cpu_destino;
	unsigned char *gpu_origen, *gpu_destino;
	struct timeval inicio, fin; 
	double tiempo;

	Mat imagen;
	imagen = imread(nombre_imagen, CV_LOAD_IMAGE_GRAYSCALE);
	Size t_imagen = imagen.size();
	alto = t_imagen.height;
	ancho = t_imagen.width;
	tamanno = sizeof(unsigned char)*alto*ancho;

	cpu_origen = (unsigned char*)malloc(tamanno);
	cpu_destino = (unsigned char*)malloc(tamanno);
	mi_error = hipMemcpyToSymbol(HIP_SYMBOL(gpu_kernel),cpu_kernel,sizeof(char)*9);
	if(mi_error != hipSuccess){printf("Error con kernel\n");exit(-1);}
	mi_error = hipMalloc((void**)&gpu_origen,tamanno);
	if(mi_error != hipSuccess){printf("Error con origen\n");exit(-1);}
	mi_error = hipMalloc((void**)&gpu_destino,tamanno);
	if(mi_error != hipSuccess){printf("Error con destino\n");exit(-1);}
	cpu_origen = imagen.data;

	gettimeofday(&inicio, NULL);
	
	mi_error = hipMemcpy(gpu_origen,cpu_origen,tamanno, hipMemcpyHostToDevice);
	if(mi_error != hipSuccess){printf("Error copiando origen primero \n");exit(-1);}
	
	int t_bloque = 32;
	dim3 dim_bloque(t_bloque,t_bloque,1);
	dim3 dim_rejilla(ceil(ancho/float(t_bloque)),ceil(alto/float(t_bloque)),1);
	filtro_sobel<<<dim_rejilla,dim_bloque>>>(gpu_origen, gpu_destino, alto, ancho);

	mi_error = hipMemcpy(cpu_destino,gpu_destino,tamanno, hipMemcpyDeviceToHost);
	if(mi_error != hipSuccess){printf("Error con \n");exit(-1);}
	gettimeofday(&fin, NULL);
	tiempo = ((fin.tv_sec  - inicio.tv_sec) * 1000000u + fin.tv_usec - inicio.tv_usec) / 1.e6;
	printf("%f\n",tiempo);		

	Mat resultado;
	resultado.create(alto,ancho,CV_8UC1);
	resultado.data = cpu_destino;
	nombre_resultado = strcat(nombre_imagen,".const.jpg");
	imwrite(nombre_resultado,resultado);
	
	//free(cpu_origen);
	//free(cpu_destino);
	hipFree(gpu_origen);
	hipFree(gpu_destino);
	
	return 0;
}
