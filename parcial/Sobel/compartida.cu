#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cv.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace cv;

__constant__ char gpu_kernel[9];

__global__ unsigned char ajustar(int valor)
{
	if(valor < 0)
	{
		valor = 0;
	}
	else if(valor > 255)
	{
		valor = 255;
	}
    return (unsigned char)valor;
}

__global__ void filtro_sobel(unsigned char *origen, unsigned char *destino, int alto, int ancho)
{
	__shared__ float chunk[1156];
	for(int a = 0; a <= 1024; a +=1024)
	{
		int posicion_chunk = threadIdx.y*32+threadIdx.x + a;
		int c_y = posicion_chunk / (34);
		int c_x = posicion_chunk % (34);
		int y = blockIdx.y * 32 + c_y - 1;
		int x = blockIdx.x * 32 + c_x - 1;
		if (x >= 0 && y >= 0 && x < ancho && y < alto)
		{
			chunk[c_y*34+c_x] = imageInput[y * ancho + x];
		}
		else
		{
			chunk[c_y*34+c_x] = 0;
		}
	}
	__syncthreads();
	int valor = 0;
	for(i = 0; i <= 2; i++)
	{
		for(j = 0; j <= 2; j++)
		{
			valor += kernel[j*3+i] chunk[(threadIdx.y+j)*34+threadIdx.x+i];
	    	}
	}
	col = blockIdx.x * 32 + threadIdx.x;
	fil = blockIdx.y * 32 + threadIdx.y;
	if (x >= 0 && y >= 0 && x < ancho && y < alto)
	destino[fil*ancho+col] = ajustar(valor);
	__syncthreads();
}

int main(int argc, char **argv)
{
	int alto, ancho, tamanno;
	char cpu_kernel[] = {-1,0,1,-2,0,2,-1,0,1}
	char *nombre_imagen = argv[1];
	char *nombre_resultado;
	nombre_resultado = (char*)malloc(sizeof(char)*255);
	unsigned char *cpu_origen, *cpu_destino;
	unsigned char *gpu_origen, *gpu_destino;
	struct timeval inicio, fin; 
	double tiempo;

	Mat imagen;
	imagen = imread(nombre_imagen, CV_LOAD_IMAGE_GRAYSCALE);
	Size t_imagen = imagen.size();
	alto = t_imagen.height;
	ancho = t_imagen.width;
	tamanno = sizeof(unsigned char)*alto*ancho;

	cpu_origen = (unsigned char*)malloc(tamanno);
	cpu_destino = (unsigned char*)malloc(tamanno);
	hipMalloc((void**)&gpu_origen,tamanno);
	hipMalloc((void**)&gpu_destino,tamanno);
	hipMemcpyToSymbol(HIP_SYMBOL(gpu_kernel),cpu_kernel,sizeof(char)*9);
	cpu_origen = imagen.data;

	gettimeofday(&inicio, NULL);
	
	hipMemcpy(gpu_origen,cpu_origen,tamanno, hipMemcpyHostToDevice);
	hipMemcpy(gpu_kernel,cpu_kernel,sizeof(char)*9, hipMemcpyHostToDevice);

	int t_bloque = 32;
	dim3 dim_bloque(t_bloque,t_bloque,1);
	dim3 dim_rejilla(ceil(ancho/float(t_bloque)),ceil(alto/float(t_bloque)),1);
	filtro_sobel<<<dim_rejilla,dim_bloque>>>(gpu_origen, gpu_destino, alto, ancho);
	hipDeviceSynchronize();

	hipMemcpy(cpu_destino,gpu_destino,tamanno, hipMemcpyDeviceToHost);

	gettimeofday(&fin, NULL);
	tiempo = ((fin.tv_sec  - inicio.tv_sec) * 1000000u + fin.tv_usec - inicio.tv_usec) / 1.e6;
	printf("%f\n",tiempo);		

	Mat resultado;
	resultado.create(alto,ancho,CV_8UC1);
	resultado.data = cpu_destino;
	nombre_resultado = strcat(nombre_imagen,".sobel_global.jpg");
	imwrite("./sobel_global.jpg",resultado);
	
	free(cpu_origen);
	free(cpu_destino);
	hipFree(gpu_origen);
	hipFree(gpu_destino);
	
	return 0;
}