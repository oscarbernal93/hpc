#include "hip/hip_runtime.h"
#include "opencv2/core/core.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include <stdio.h>
#include <stdlib.h>
#include <cv.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

using namespace cv;

__constant__ char gpu_kernel[9];

__device__ unsigned char ajustar(int valor)
{
	if(valor < 0)
	{
		valor = 0;
	}
	else if(valor > 255)
	{
		valor = 255;
	}
    return (unsigned char)valor;
}

__global__ void filtro_sobel(unsigned char *origen, unsigned char *destino, int alto, int ancho)
{
	__shared__ float chunk[1156];
	int posicion_chunk,c_y,c_x,y,x,i,j,col,fil;
	for(int a = 0; a <= 1024; a +=1024)
	{
		posicion_chunk = threadIdx.y*32+threadIdx.x + a;
		c_y = posicion_chunk / (34);
		c_x = posicion_chunk % (34);
		y = blockIdx.y * 32 + c_y - 1;
		x = blockIdx.x * 32 + c_x - 1;
		if(c_x < 34 && c_y < 34){
			if (x >= 0 && y >= 0 && x < ancho && y < alto)
			{
				chunk[c_y*34+c_x] = origen[y * ancho + x];
			}
			else
			{
				chunk[c_y*34+c_x] = 0;
			}
		}
	}
	__syncthreads();
	int valor = 0;
	for(i = 0; i <= 2; i++)
	{
		for(j = 0; j <= 2; j++)
		{
			valor += gpu_kernel[j*3+i]*chunk[(threadIdx.y+j)*34+threadIdx.x+i];
	    	}
	}
	col = blockIdx.x * 32 + threadIdx.x;
	fil = blockIdx.y * 32 + threadIdx.y;
	if (x >= 0 && y >= 0 && x < ancho && y < alto)
		destino[fil*ancho+col] = ajustar(valor);
	__syncthreads();
}

int main(int argc, char **argv)
{
	hipError_t mi_error = hipSuccess;
	int alto, ancho, tamanno;
	char cpu_kernel[] = {-1,0,1,-2,0,2,-1,0,1};
	char *nombre_imagen = argv[1];
	char *nombre_resultado;
	nombre_resultado = (char*)malloc(sizeof(char)*255);
	unsigned char *cpu_origen, *cpu_destino;
	unsigned char *gpu_origen, *gpu_destino;
	struct timeval inicio, fin; 
	double tiempo;

	Mat imagen;
	imagen = imread(nombre_imagen, CV_LOAD_IMAGE_GRAYSCALE);
	Size t_imagen = imagen.size();
	alto = t_imagen.height;
	ancho = t_imagen.width;
	tamanno = sizeof(unsigned char)*alto*ancho;

	cpu_origen = (unsigned char*)malloc(tamanno);
	cpu_destino = (unsigned char*)malloc(tamanno);
	mi_error = hipMemcpyToSymbol(HIP_SYMBOL(gpu_kernel),cpu_kernel,sizeof(char)*9);
	if(mi_error != hipSuccess){printf("Error con kernel\n");exit(-1);}
	mi_error = hipMalloc((void**)&gpu_origen,tamanno);
	if(mi_error != hipSuccess){printf("Error con origen\n");exit(-1);}
	mi_error = hipMalloc((void**)&gpu_destino,tamanno);
	if(mi_error != hipSuccess){printf("Error con destino\n");exit(-1);}
	cpu_origen = imagen.data;

	gettimeofday(&inicio, NULL);
	
	mi_error = hipMemcpy(gpu_origen,cpu_origen,tamanno, hipMemcpyHostToDevice);
	if(mi_error != hipSuccess){printf("Error copiando origen\n");exit(-1);}

	int t_bloque = 32;	
	dim3 dim_bloque(t_bloque,t_bloque,1);
	dim3 dim_rejilla(ceil(ancho/float(t_bloque)),ceil(alto/float(t_bloque)),1);
	filtro_sobel<<<dim_rejilla,dim_bloque>>>(gpu_origen, gpu_destino, alto, ancho);
	hipDeviceSynchronize();

	mi_error = hipMemcpy(cpu_destino,gpu_destino,tamanno, hipMemcpyDeviceToHost);
	if(mi_error != hipSuccess){printf("Error copiando destino: %s\n",hipGetErrorString(mi_error));exit(-1);}
	gettimeofday(&fin, NULL);
	tiempo = ((fin.tv_sec  - inicio.tv_sec) * 1000000u + fin.tv_usec - inicio.tv_usec) / 1.e6;
	printf("%f\n",tiempo);		

	Mat resultado;
	resultado.create(alto,ancho,CV_8UC1);
	resultado.data = cpu_destino;
	nombre_resultado = strcat(nombre_imagen,".comp.jpg");
	imwrite(nombre_resultado,resultado);
	
	//free(cpu_origen);
	//free(cpu_destino);
	hipFree(gpu_origen);
	hipFree(gpu_destino);
	
	return 0;
}
